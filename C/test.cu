#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void sleep_kernel(float seconds) {
    unsigned long long start_clock = clock64();
    unsigned long long wait_clocks = (unsigned long long)(seconds * 1.0e9);

    while (clock64() - start_clock < wait_clocks) {
        // do nothing, just wait
    }
}

int main(int argc, char const *argv[]) {
	float sleep_time = atof(argv[1]); // in seconds
    std::cout << "Sleeping for " << sleep_time / 60 << " minutes..." << std::endl;
  
    sleep_kernel<<<1, 1>>>(sleep_time);
    hipDeviceSynchronize();

    std::cout << "Woke up after " << sleep_time / 60 << " minutes!" << std::endl;
    return 0;
}